#include "hip/hip_runtime.h"
/*
	Code copied from SETH's HOME PAGE[http://www-personal.umich.edu/~smeyer/cuda/grid.pdf] and edited by Quswar[CS2003]
*/

#include <stdio.h>

#define N 16

__global__ void matrixMult (int *a, int *b, int *c, int width) {
 int k, sum = 0;
 int col = threadIdx.x + blockDim.x * blockIdx.x;
 int row = threadIdx.y + blockDim.y * blockIdx.y;
 if(col < width && row < width) {
 for (k = 0; k < width; k++)
 sum += a[row * width + k] * b[k * width + col];
 c[row * width + col] = sum;
 }
}

int main() {
 int h_a[N][N], h_b[N][N], h_c[N][N];
 int *d_a, *d_b, *d_c;
 
 //initialize a and b with real values
 for (int i=0; i<N; i++){
        for (int j=0; j<N; j++){
                h_a[i][j] = 2;//(i*j)+i;
                h_b[i][j] = 1;//(i*j)+i;
        }
 }

 
 int size = N * N * sizeof(int);
 
 hipMalloc((void **) &d_a, size);
 hipMalloc((void **) &d_b, size);
 hipMalloc((void **) &d_c, size);
 
 hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
 hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice);
 
 dim3 dimGrid(1, 1);
 dim3 dimBlock(N, N);
 
 matrixMult<<<dimGrid, dimBlock>>>(d_a, d_b, d_c, N);
 
 hipMemcpy(h_c, d_c, size, hipMemcpyDeviceToHost);
 
 hipFree(dev_a); hipFree(dev_b); hipFree(dev_c);
 
 //print output
 for (int i=0; i<N; i++){
        for (int j=0; j<N; j++){
                printf("%d  ", h_c[i][j]);
        }
        printf("\n");
 }


}