/*
	Code copied from SETH's HOME PAGE[http://www-personal.umich.edu/~smeyer/cuda/grid.pdf] and edited by Quswar[CS2003]
*/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#define N 32
#define BLOCK_DIM 32

__global__ void matrixAdd (int *a, int *b, int *c) {
 int col = blockIdx.x * blockDim.x + threadIdx.x;
 int row = blockIdx.y * blockDim.y + threadIdx.y;
 int index = col + row * N;
 if (col < N && row < N) {
 c[index] = a[index] + b[index];
 }
}



int main() {
 int h_a[N][N], h_b[N][N], h_c[N][N];
 int *d_a, *d_b, *d_c;
 int size = N * N * sizeof(int);

 //initialize a and b with real values
 for (int i=0; i<N; i++){
        for (int j=0; j<N; j++){
                h_a[i][j] = 1;
                h_b[i][j] = 0;
        }
 }

 hipMalloc((void**)&d_a, size);
 hipMalloc((void**)&d_b, size);
 hipMalloc((void**)&d_c, size);

 hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
 hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice);

 dim3 dimBlock(BLOCK_DIM, BLOCK_DIM);
 dim3 dimGrid((int)ceil(N/dimBlock.x),(int)ceil(N/dimBlock.y));

 matrixAdd<<<dimGrid,dimBlock>>>(d_a,d_b,d_c);

 hipMemcpy(h_c, d_c, size, hipMemcpyDeviceToHost);

 /*free(h_a); free(h_b); free(h_c);*/
 hipFree(d_a); hipFree(d_b); hipFree(d_c);

 //print output
 for (int i=0; i<N; i++){
        for (int j=0; j<N; j++){
                printf("%d  ", h_c[i][j]);
        }
        printf("\n");
 }

}
